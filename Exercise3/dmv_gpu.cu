#include "hip/hip_runtime.h"
/*
 *  dmv_gpu.cu -- Template for DMV GPU kernels
 *
 *  Copyright (C) 2010-2013, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2010-2013, Vasileios Karakasis
 */ 
#include <stdio.h>
#include "dmv.h"

/*
 *  Utility function to get the thread ID within the
 *  global working space.
 */ 
__device__ int get_global_tid()
{
    return (gridDim.x*blockIdx.y + blockIdx.x)*blockDim.x*blockDim.y +
        blockDim.x*threadIdx.y + threadIdx.x;
}

/*
 *  Utility function to get the thread ID within the
 *  local/block working space.
 */ 
__device__ int get_local_tid()
{
    return blockDim.x*threadIdx.y + threadIdx.x;
}

/*
 *  Naive kernel
 */ 
__global__ void dmv_gpu_naive(const value_t *a, const value_t *x, value_t *y,
                              size_t n)
{
	int i;
	int idx = get_global_tid();

    if (idx >= n) {
        return;
    }
	
	y[idx] = 0;
    for (i = 0; i < n; ++i) {
        y[idx] += a[idx*n+i]*x[i];
	}
        
}

/*
 *  Coalesced memory acceses
 */
__global__ void dmv_gpu_coalesced(const value_t *a, const value_t *x,
                                  value_t *y, size_t n)
{
    int i, j;
	int idx = get_global_tid();

    if (idx >= n) {
        return;
    }
	
	y[idx] = 0;
    for (i = 0; i < n; ++i) {
        y[idx] += a[idx*n+i]*x[i];
	} 
}

/*
 *  Use of shared memory
 */
__global__ void dmv_gpu_shmem(const value_t *a, const value_t *x, value_t *y,
                              size_t n)
{
	int i,j;
	int idx = get_global_tid();
	int idx2 = get_local_tid();
	extern __shared__ value_t shmem[];
	
	if (idx >= n) {
        return;
    }
	
	y[idx] = 0;
	for(j = idx2; j < n; j += blockDim.y) {
        shmem[j] = x[j];
        __syncthreads();
        for(i = j-idx2; i < j-idx2+blockDim.y; i++) {
                y[idx] += a[i*n+idx]*shmem[i];
        }
    }	
}
